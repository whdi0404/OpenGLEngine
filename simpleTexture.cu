#include "hip/hip_runtime.h"
#include "simpleTexture.cuh"

__global__ void transformKernel(float *outputData,
                                int width,
                                int height,
                                float theta)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = (float)x - (float)width/2; 
    float v = (float)y - (float)height/2; 
    float tu = u*cosf(theta) - v*sinf(theta); 
    float tv = v*cosf(theta) + u*sinf(theta); 

    tu /= (float)width; 
    tv /= (float)height; 

    // read from texture and write to global memory
    outputData[y*width + x] = tex2D(tex, tu+0.5f, tv+0.5f);
}
extern "C"
void runTest(void* texData, int width, int height, int textureDataSize)
{
    // Allocate device memory for result
    float *dData = NULL;
    checkCudaErrors(hipMalloc((void **) &dData, textureDataSize));

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,
                                    &channelDesc,
                                    width,
                                    height));
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      texData,
									  textureDataSize,
                                      hipMemcpyHostToDevice));

    // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Warmup
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle);

    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",
           (width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // Allocate mem for the result on host side
    char *hOutputData = (char *) malloc(textureDataSize);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData,
                               dData,
							   textureDataSize,
                               hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
}

__global__ void increment_kernel(int *g_data, int inc_value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	g_data[idx] = g_data[idx] + inc_value;
}

extern "C"
int RunKernel()
{
	int n = 16 * 1024 * 1024;
	int nbytes = n * sizeof(int);
	int value = 26;

	// allocate host memory
	int *a = 0;
	checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
	memset(a, 0, nbytes);

	// allocate device memory
	int *d_a = 0;
	checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
	checkCudaErrors(hipMemset(d_a, 255, nbytes));

	// set kernel launch configuration
	dim3 threads = dim3(512, 1);
	dim3 blocks = dim3(n / threads.x, 1);

	// create cuda event handles
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);

	checkCudaErrors(hipDeviceSynchronize());
	float gpu_time = 0.0f;

	// asynchronously issue work to the GPU (all to stream 0)
	sdkStartTimer(&timer);
	hipEventRecord(start, 0);
	hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
	increment_kernel<<<blocks, threads, 0, 0 >>>(d_a, value);
	hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
	hipEventRecord(stop, 0);
	sdkStopTimer(&timer);

	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter = 0;

	while (hipEventQuery(stop) == hipErrorNotReady)
	{
		counter++;
	}

	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f\n", gpu_time);
	printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
	printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

	// check the output for correctness

	// release resources
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipFree(d_a));

	return 0;
}
